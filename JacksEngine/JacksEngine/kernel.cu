#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>
#include "Defs.cuh"
#include "UCI.cuh"
#include "Bitboards.cuh"
#include <ctime>

int main()
{
	printf("Welcome to Jack's Chess Engine \n");

	InitAllBitBoard();
	
	U64 playBitBoard = 0ULL;

	printf("Start: \n");

	PrintBitboard(playBitBoard);

	playBitBoard |= (1ULL << SQ64(D2));
	playBitBoard |= (1ULL << SQ64(D3));
	playBitBoard |= (1ULL << SQ64(D4));
	playBitBoard |= (1ULL << SQ64(D1));

	printf("D2 added: \n\n");
	PrintBitboard(playBitBoard);
	auto count = CNT(playBitBoard);

	printf("Count: %d \n", count);

	auto index = POP(&playBitBoard);

	PrintBitboard(playBitBoard);
	count = CNT(playBitBoard);

	printf("Count: %d \n", count);


	UCI::loop();

    return 0;
}



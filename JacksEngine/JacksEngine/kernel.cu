#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>
#include "UCI.cuh"


__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
	printf("ID : %d\n", threadIdx.x);
    c[i] = a[i] + b[i];
}

int main()
{
	printf("Welcome to Jack's Chess Engine \n");

	UCI::loop();

    return 0;
}




#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>
#include "Defs.cuh"
#include "UCI.cuh"

int main()
{
	printf("Welcome to Jack's Chess Engine \n");

	InitSq120To64();
	
	U64 playBitBoard = 0ULL;

	printf("Start: \n");

	PrintBitboard(playBitBoard);

	playBitBoard |= (1ULL << SQ64(D2));

	printf("D2 added: \n\n");
	PrintBitboard(playBitBoard);

	UCI::loop();

    return 0;
}



#include "hip/hip_runtime.h"
#include "UCI.cuh"
#include <ostream>
#include <iostream>
#include <string>
#include <sstream>
#include <>
#include <hip/hip_runtime_api.h>

using namespace std;

__global__ void testIndex()
{
	printf("Block id : %d Thread id : %d\n", blockIdx.x, threadIdx.x);
}

void UCI::loop()
{
	string token, cmd;
	do
	{
		if (!getline(cin, cmd)) // Block here waiting for input or EOF
			cmd = "quit";

		istringstream is(cmd);

		token.clear(); // getline() could return empty or blank line
		is >> skipws >> token;

		// The GUI sends 'ponderhit' to tell us to ponder on the same move the
		// opponent has played. In case Signals.stopOnPonderhit is set we are
		// waiting for 'ponderhit' to stop the search (for instance because we
		// already ran out of time), otherwise we should continue searching but
		// switching from pondering to normal search.
		if (token == "quit"
			|| token == "stop"
			|| (token == "ponderhit"))
		{
			cout << "Im out of here";
		}

		else if (token == "uci")
			cout << "id name " << "Jacks engine"
			<< "\nuciok" << endl;

		else if (token == "ucinewgame")
		{
			//Search::clear();
			//Tablebases::init(Options["SyzygyPath"]);
			//Time.availableNodes = 0;
		}
		else if (token == "testCudas")
		{
			testIndex <<< 1, 1024 >>>();
			hipDeviceSynchronize();
		}
		else if (token == "isready") cout << "readyok" << endl;
		else if (token == "go") {
			//	go(pos, is);
			cout << "Lets roll" << endl;
		}

		else
			cout << "Unknown command: " << cmd << endl;
	} while (token != "quit");
}
